#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

typedef unsigned char byte;
texture<byte ,hipTextureType1D,hipReadModeNormalizedFloat> texSrc;

__global__ void CUDATex(float* dst){
 int x = threadIdx.x;
 int y = blockIdx.x;
 int w = blockDim.x;
 int index = y*w+x;
 dst[index] = tex1D(texSrc, (float)index/(w-1));
}

int main()
{ 
	int w = 4;
	int dstW = 8;
	byte *src_h = new byte[w];
	float *out_h = new float[dstW];
	for (int i = 0; i < w; i++)
	{
	src_h[i] = i * 85;
	printf("host  in[%d] %d\n",i, src_h[i]);
	}

	float *src_d;
	hipArray * cu_array; 
	hipMalloc(&src_d, dstW*sizeof(float)); 
	hipMallocArray(&cu_array, &texSrc.channelDesc, w);
	texSrc.filterMode = hipFilterModeLinear;
	texSrc.addressMode[0] = hipAddressModeClamp;
	texSrc.normalized = true;
	hipBindTextureToArray(texSrc, cu_array, texSrc.channelDesc);
	hipMemcpyToArray(cu_array, 0, 0, src_h, w, hipMemcpyHostToDevice);  

	printf("normalize On GPU \n");
	CUDATex<<<1,dstW>>>(src_d);
	hipMemcpy(out_h,src_d, dstW*sizeof(float), hipMemcpyDeviceToHost); 
		
	for (int i = 0; i < dstW; i++)
	{
		printf("host out[%d] %f\n",i, out_h[i]);
	}

    return 0;
}