#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"#include ""
#include <stdio.h>
#include <malloc.h>
#include <hiprand.h>

__global__ void mat_sum_by_row(
	float* dst, float*src,int stride){
	int tx = threadIdx.x;
	float sum = 0;
	for (int i = 0; i < stride; i++)	
		sum += src[tx + i * stride];	
	dst[tx] = sum;
}
__global__ void mat_sum_by_column(
	float* dst, float*src, int stride){
	int tx = threadIdx.x;
	float sum = 0;
	for (int i = 0; i < stride; i++)
		sum += src[tx * stride + i]; //���� : [tx + i * stride]
	dst[tx] = sum;
}
void matrix_sum_by_row(){
	int h = 4, w = 4;
	float *src_h, *dst_h;
	float *src_d, *dst_d;
	src_h = (float*)malloc(h*w*sizeof(float));
	dst_h = (float*)malloc(w*sizeof(float));
	hipMalloc(&src_d, h*w*sizeof(float));
	hipMalloc(&dst_d, w*sizeof(float));
	for (int i = 0; i < h*w; i++) src_h[i] = i;
	hipMemcpy(src_d, src_h, h*w*sizeof(float), hipMemcpyHostToDevice);
	mat_sum_by_column << <1, w >> > (dst_d, src_d, w);
	hipMemcpy(dst_h, dst_d, w*sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < w; i++)
		printf("%d %f\n", i, dst_h[i]);
}
//__global__������ ȣ���ϴ� �Լ�
__device__ float multi(float a, float b){
	return a * b;
}

const int W = 6;
__global__ void filter_1d_kernel(float*dst, float*src, float*filter, int f){
	int tx = threadIdx.x;//0,1,2,3
	__shared__ float shared_memory[W];//��� ���� �����˴ϴ�
	shared_memory[tx] = src[tx];// global memory�� �����͸� �����޸𸮷� ����
	if (tx > 1){
		shared_memory[tx+2] = src[tx+2];
	}
	__syncthreads();//����ȭ : ���� ��ϳ��� ��� �����尡 �۾��� ��ĥ������ ����ض�
	float sum = 0; // on-chip �ӵ� ����	
	for (int i = 0; i < f; i++)//f=3
	{	
		sum += multi(shared_memory[tx + i], filter[i]);
	}
	dst[tx] = sum;
}
void filter_1d(){
	hipDeviceSynchronize();// gpu ���� ����ȭ
	int w = 6, f = 3;
	int out_length = w - (f / 2) * 2;// 4
	float *src_h, *filter_h, *dst_h;
	float *src_d, *filter_d, *dst_d;
	src_h = (float*)malloc(w*sizeof(float));
	filter_h = (float*)malloc(f*sizeof(float));
	dst_h = (float*)malloc(out_length*sizeof(float));
	hipMalloc(&src_d, w*sizeof(float));
	hipMalloc(&filter_d, f*sizeof(float));
	hipMalloc(&dst_d, out_length*sizeof(float));

	for (int i = 0; i < w; i++) src_h[i] = i;//�Է� ��ȣ
	for (int i = 0; i < f; i++) filter_h[i] = 1;//���� ��� 
	hipMemcpy(src_d, src_h, w*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(filter_d, filter_h, f*sizeof(float), hipMemcpyHostToDevice);
	filter_1d_kernel <<<1, out_length>>>(dst_d, src_d, filter_d, f);
	hipMemcpy(dst_h, dst_d, out_length*sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < out_length; i++)
		printf("%d %f\n", i, dst_h[i]);
}
// f=3, w=6, out_w=4
__global__ void filter_2d_kernel(float*dst, float*src, float*filter, int f, int w, int out_w){
	int bx = blockIdx.x; //0, 1
	int tx = threadIdx.x;//0,1,2,3
	// src (2, 6) �� 12���� ������
	//__shared__ float shared_memory[W];//���� ���� �޸�
	//__shared__ int anohter_memory[W];//���� ���� �޸�
	extern __shared__ float shared_memory[];//���� ���� �޸� // 2*6*4(byte)
	//float * first_shared = shared_memory;
	//int * another_shared = (int*)&shared_memory[6];

	// 0�� ��� ������ 0,1,2,3 : src[0~5]  �� �����͸� ���0�� �����޸𸮷� ����
	// 1�� ��� ������ 0,1,2,3 : src[6~11] �� �����͸� ���1�� �����޸𸮷� ����
	shared_memory[tx] = src[bx * W + tx]; //src[0~3], src[6~9] ���� �Ϸ�
	if (tx > 1){
		// 0�� ����� ������ 2�� src[4] �� �����޸�[4] �� �Űܶ�. 
		shared_memory[tx + 2] = src[bx * W + tx + 2];
	}
	__syncthreads();//��� ���� ���� ����ȭ
	float sum = 0;
	for (int i = 0; i < f; i++)//f=3
	{			
		sum += shared_memory[tx + i] * filter[i];
	}
	// 0���/0������� (0)�� ���� ����, 1���/0��������� (4) �� ���� ����Ѵ�
	dst[bx * out_w + tx] = sum;
}

void checkCudaErrors(hipError_t error){
	if (error != hipError_t::hipSuccess)
		printf("error : %d %s \n", error, hipGetErrorString(hipGetLastError()));
}
void cudaCheck(hipError_t error){
	checkCudaErrors(error);
}

void filter_2d(){
	int w = 6, h = 2, f = 3;//��ȣ�� ���� 6, ��ȣ�� ���� 2�� 
	int out_length = h * (w - (f / 2) * 2);// 2 * 4
	float *src_h, *filter_h, *dst_h;
	float *src_d, *filter_d, *dst_d;
	src_h = (float*)malloc(h * w*sizeof(float));
	filter_h = (float*)malloc(f*sizeof(float));
	dst_h = (float*)malloc(out_length*sizeof(float));
	checkCudaErrors(hipMalloc(&src_d, h * w*sizeof(float)));
	checkCudaErrors(hipMalloc(&filter_d, f*sizeof(float)));
	checkCudaErrors(hipMalloc(&dst_d, out_length*sizeof(float)));	

	for (int i = 0; i < h * w; i++) src_h[i] = i;//�Է� ��ȣ
	for (int i = 0; i < f; i++) filter_h[i] = 1;//���� ��� 
	hipMemcpy(src_d, src_h, h * w*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(filter_d, filter_h, f*sizeof(float), hipMemcpyHostToDevice);
	// <<< ��ϼ�, �������, �����޸� >>>
	filter_2d_kernel <<<2, 4, 2*W*sizeof(float) >>>(dst_d, src_d, filter_d, f, w, 4);
	hipMemcpy(dst_h, dst_d, out_length*sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < out_length; i++)
		printf("%d %f\n", i, dst_h[i]);
}

__device__ int globalVar = 10;
__constant__ int a_gpu = 1; // GPU���� ����ϴ� ���, ĳ�� ����, 64kb ����
__constant__ int k_gpu[] = { 1, 2, 3, 4, 5 }; //�����Ҵ�
const int a = 1; //��� 
const int a_[5] = { 1, 2, 3, 4, 5 };


__global__ void hello_kernel(int* src){
	// ĳ�ð� �Ǹ� �޸� > ĳ�� > �ھ�� �о���� �ܰ踦 ���� �� �ֽ��ϴ�.  
	int v = k_gpu[0]; //��� �޸𸮴� __global__ ���� �ٷ� ����� �� �ֽ��ϴ�. 
}
// __global__ : gpu �Լ��ε�, host���� ȣ��
// __host__ : host �Լ��Դϴ�. ������� ������ �ڵ����� �߰��˴ϴ�. 
// __device__ : device���� ȣ��
// __host__ __device__ : ���ʿ��� ��� ���� 
// __constant__ : ��� �޸�

__host__ void device_query(){
	//multi-gpu �϶� ���� �ٸ� ����� �����Ҷ� ����մϴ�
	int count = 0;
	hipGetDevice(&count); // ���� ���� �����ִ� gpu ���?
	hipGetDeviceCount(&count); //gpu ���� 
	printf("gpu count : %d\n", count);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0); // 0�� gpu �Ӽ� ��������
	printf("prop.totalGlobalMem : %d \n", prop.totalGlobalMem);// �޸� ũ��
	printf("prop.multiProcessorCount : %d \n", prop.multiProcessorCount);// SM 8��
	printf("prop.totalConstMem: %d \n", prop.totalConstMem);
	
	
	int *gpu_0, *gpu_1;
	
	hipSetDevice(0); // gpu 0���� ����ϰڴ�. ������ ��� ����� gpu 0������ ����˴ϴ�
	hipMalloc(&gpu_0, 100); // gpu 0 �� �Ҵ�
	hello_kernel << <1, 1 >> >(gpu_0); // gpu 0���� ����
	
	hipMalloc(&gpu_1, 100); // gpu 1 �� �Ҵ�
	hipSetDevice(1);
	hello_kernel << <1, 1 >> >(gpu_1); // gpu 1���� ���� ����
}

__device__ float mean(float2 src){
	// __ �Լ� : Intrinsics �ӵ��� ������ ���е��� �ణ �������ϴ� : Fast math �Լ�
	//return sqrt(pow(src.x, src.y)) + __cosf(src.x); //cuda math
	//return (src.x + src.y) * 0.5;
	double a = 3.0;
	return __fadd_rn(src.x, src.y) * 0.5;
	//Thrust ���� ���� ����, 
}

__global__ void channel_mean_kernel(float *dst, float2 *src){
	//<< < m, dim3(w, h, z) >> >  h * w * z <= 1024
	int bx = blockIdx.x;  // 0, 1
	int ty = threadIdx.y; // 0, 1
	int tx = threadIdx.x; // 0, 1, 2, 3 
	
	// 0~15, 8:����� ������ ����, 4:����� �� ���� ������ ����
	int index = (bx * blockDim.x * blockDim.y) + (ty * blockDim.x) + tx;
	register int a = 10;//register �޸�(�ѵ� �ʰ��� �ڵ����� local memory ���)
	int b = 10;
	int temp[25];// ���� �迭 ���� : �׷��� ������ �ʽ��ϴ�. local memory
	int *temp2;// ���� �迭 ���� local memory
	// new, malloc ����
	temp2 = (int*)malloc(100); // �ſ� �����ϴ�,�ʿ��� ��ũ��ġ ���۸�hipMalloc �ؼ� ���ڷ� ����  
	free(temp2);
	temp2[index] = dst[index];
	temp[index] = dst[index]; // ������ ���� �ʴ� �ڵ�� �����Ϸ��� �����մϴ� 
	dst[index] = mean(src[index]) + temp[index] + temp2[index];
}

__host__ void channel_mean(){
	int m = 2, h = 2, w = 4;
	//uint3
	float2 *src_h, *src_d;
	float *dst_h, *dst_d;
	src_h = (float2*)malloc(m * h * w * sizeof(float2));
	dst_h = (float*)malloc(m * h * w * sizeof(float));	
	checkCudaErrors(hipMalloc(&src_d, m * h * w * sizeof(float2)));
	checkCudaErrors(hipMalloc(&dst_d, m * h * w * sizeof(float)));
	for (int i = 0; i < m * h * w; i++)
	{
		src_h[i].x = i * 2;
		src_h[i].y = i * 2 + 1;
	}
	
	hipMemcpy(src_d, src_h, m * h * w * sizeof(float2), hipMemcpyHostToDevice);	
	//����
	hipEvent_t start, stop;//����ü ����, �������� �����ؼ� ����ϼ���
	hipEventCreate(&start);//����ü �ʱ�ȭ
	hipEventCreate(&stop);
	hipEventRecord(start);//���� �ð� ���
	channel_mean_kernel <<< m,dim3(w, h, 1)>>>(dst_d, src_d);
	hipEventRecord(stop);//�� �ð� ���
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);// ���� ��������
	printf("elapsedTime : %f ms \n", elapsedTime);
	
	hipMemcpy(dst_h, dst_d, m * h * w * sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < m * h * w; i++)
		printf("%d %f \n", i, dst_h[i]);
}

__global__ void max_kernel(float *dst, float *src){
	int tx = threadIdx.x;
	//src[tx];
	printf("%f %f \n", dst[0], src[tx]);
	//float atomicAdd(float *address ���� ������ �ּ�, float val ��)	
	float old = atomicAdd(dst, src[tx]); //�ӵ��� ���� �������� �ǿܷ� �����ϴ�.	
	printf("%f %f \n", old, dst[0]);
}
//src�� 16�� ���� �迭
__global__ void sum_kernel(float *dst, float *src){
	int tx = threadIdx.x;	
	extern __shared__ float sm[ ];	
	sm[tx] = src[tx];//���� ������ 1���� �ű�ϴ�. 
	__syncthreads();// �����Ͱ� ���� ����ɶ����� ��Ϻ��� ���
	//blockDim.x = 16
	for (int i = 1; i < blockDim.x; i *= 2){//i = { 1, 2, 4, 8, 16(x)}
		if (tx % 2 * i == 0){ // {2*1�� ���, 2*2 �� ���, 2*4 �� ���, 2*8�� ���
			sm[tx] = sm[tx] + sm[tx + i];	//A �۾��� �ϴ� ������ ����[0~31][32~63]		
		}
		else{
			// B �۾��� �ϴ� ������ ����
		}
	}
	if (tx == 0)
		dst[tx] = sm[tx];
}

__host__ void atomic_func(){
	// ���� ����, ��Ƽ ������ ȯ�濡�� race condition(���� ����) ������ ���ϱ� ���� ���
	// sum, max, min ���� �۾��� �Ҷ� ��������� ���������� �۾��� �� �� �ְ� ���ݴϴ�

	int size = 16;
	float *src_h, *src_d;
	float sum_h = 0, *sum_d;
	src_h = (float*)malloc(size * sizeof(float));	
	hipMalloc(&src_d, size * sizeof(float));
	hipMalloc(&sum_d, 1 * sizeof(float));
	for (int i = 0; i < size; i++) src_h[i] = i;//{0,1,2,3,4}
	hipMemcpy(src_d, src_h, size * sizeof(float), hipMemcpyHostToDevice);
	hipMemset(sum_d, 0, sizeof(float));
	sum_kernel << <1, size, size*sizeof(float) >> >(sum_d, src_d);
	hipMemcpy(&sum_h, sum_d, sizeof(float), hipMemcpyDeviceToHost);
	printf("sum : %f \n", sum_h);
}

void check_curand(hiprandStatus_t status){
	printf("status %d \n", status);
}
void hiprand(){
	size_t n = 100;
	hiprandGenerator_t gen;
	float *devData, *hostData;
	/* Allocate n floats on host */
	hostData = (float *)calloc(n, sizeof(float));
	/* Allocate n floats on device */
	hipMalloc((void **)&devData, n * sizeof(float));
		
	check_curand(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));	
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
	/* Generate n floats on device */
	hiprandGenerateUniform(gen, devData, n);
	//hiprandGenerateNormal(gen, devData, n, 0.0f, 1.0f);
	/* Copy device memory to host */
	hipMemcpy(hostData, devData, n * sizeof(float), hipMemcpyDeviceToHost);
	/* Show result */
	for (int i = 0; i < n; i++) {
		printf("%1.4f ", hostData[i]);
	}
	printf("\n");

}

void extern_call(char* src){
	hipHostRegister(&src, 100, hipHostRegisterDefault);
}

void pinned_memory(){
	int size = 10000000; // 10Mb
	char * src_h = (char*)malloc(size);//�Ϲ� �޸�
	char * src_h_pin;// ������ �޸�: raw �����Ϳ� ����մϴ�
	hipHostAlloc(&src_h_pin, size, hipHostMallocMapped);

	char * gpu;
	hipMalloc(&gpu, size);
	hipMemcpy(gpu, src_h, size, hipMemcpyHostToDevice); 
	hipMemcpy(gpu, src_h_pin, size, hipMemcpyHostToDevice);

	hipMemcpy(src_h, gpu, size, hipMemcpyDeviceToHost);
	hipMemcpy(src_h_pin, gpu, size, hipMemcpyDeviceToHost);
}

__global__ void MyKernel(float* dst, float* src, int size){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	dst[index] = cos(src[index]) + sin(src[index]);
}
void stream(){	//3.2.5.5. Streams
	int size = 100 * 512; 
	hipStream_t stream[2]; // ����
	for (int i = 0; i < 2; ++i)
		hipStreamCreate(&stream[i]); //�ʱ�ȭ
	float* hostPtr;
	hipHostMalloc(&hostPtr, 2 * size);// malloc �� ����
	float * inputDevPtr, *outputDevPtr;
	hipMalloc(&inputDevPtr, 2 * size);
	hipMalloc(&outputDevPtr, 2 * size);

	for (int i = 0; i < 2; ++i) {
		hipMemcpyAsync(inputDevPtr + i * size, hostPtr + i * size,
			size, hipMemcpyHostToDevice, stream[i]);
		MyKernel << <100, 512, 0, stream[i] >> >
			(outputDevPtr + i * size, inputDevPtr + i * size, size);
		hipMemcpyAsync(hostPtr + i * size, outputDevPtr + i * size,
			size, hipMemcpyDeviceToHost, stream[i]);
	}
	//ù��° ���� ���Ŀ��� ��Ʈ���� ���� �ʿ䰡 �����ϴ�.(ī�ǰ� �����ϱ�)
	//����ȭ�� �ʿ�����ϴ�. : default stream���� (����ȭ�� �ɸ� �Ŀ�) ����˴ϴ�. 
	MyKernel << <200, 512>> > (outputDevPtr, inputDevPtr, size);
}

const int m = 1000;
struct AOS{
	float a[30];
	float b[30];
	float c[30];
};
struct SOA{// ����ü�� ��� ���� �ʴ� ���̳� ��������
	float a[30][m];
	float b[30][m];
	float c[30][m];
};
__global__ void AOS_function(AOS *aoss){//4 �� ���� ~ 20 �� ������
	int tx = threadIdx.x;//10���� ������ 
	AOS aos = aoss[tx];// ������ �ϳ��� ����ü �ϳ��� ����ؼ� �۾�
	int sum = aos.a[0] + aos.a[1] + aos.a[2];
	aos.c[0] = sum;
}
__global__ void SOA_function(SOA *soa){
	int tx = threadIdx.x;//10���� ������
	int sum = soa->a[0][tx] + soa->a[1][tx] + soa->a[2][tx];
	soa->c[0][tx] = sum;
}
void data_layout(){	
	AOS aos[m]; // CPU ���� ȿ����
	SOA soa; // GPU ���� ȿ����
	AOS *aos_d;
	SOA *soa_d;
	hipMalloc(&aos_d, m * sizeof(AOS));
	hipMalloc(&soa_d, sizeof(SOA));
	AOS_function << <1, m >> >(aos_d);
	SOA_function << <1, m >> >(soa_d);
}

#include <npp.h>
typedef unsigned char uchar;
void nppFloatSum()
{
	const int w = 2;
	const int h = 3;
	const int arraySize = w * h;
	const float b[arraySize] = { 0, 10, 20, 30, 40, 50 };
	float* b_d;
	float* pSum;
	float nSumHost;
	hipMalloc(&b_d, sizeof(float)* arraySize);
	hipMalloc((void **)(&pSum), sizeof(float));
	hipMemcpy(b_d, b, sizeof(float)* arraySize, hipMemcpyHostToDevice);
	uchar * pDeviceBuffer;

	int nBufferSize;
	nppsSumGetBufferSize_32f(arraySize, &nBufferSize);
	printf("nppsSumGetBufferSize_32f = %d\n", nBufferSize);
	// Allocate the scratch buffer
	hipMalloc((void **)(&pDeviceBuffer), nBufferSize);
	nppsSum_32f(b_d, arraySize, pSum, pDeviceBuffer);
	hipMemcpy(&nSumHost, pSum, sizeof(float), hipMemcpyDeviceToHost);
	printf("float sum = %f\n", nSumHost);
}
#include <hipfft/hipfft.h>
void fft(){
	//cufft.lib ��Ŀ �߰�
	int length = 1000;
	float2 * src = (float2*)malloc(length*sizeof(float2));
	for (int i = 0; i < length; i++)
	{
		src[i].x = i;//�Ǽ�
		src[i].y = 0;//���
	}
	float2 *src_d;
	hipMalloc(&src_d, length*sizeof(float2));
	hipMemcpy(src_d, src, length*sizeof(float2), hipMemcpyHostToDevice);

	hipfftHandle plan;
	hipfftPlan1d(&plan, length, HIPFFT_C2C, 1);//�Ķ���� ����
	hipfftExecC2C(plan, src_d, src_d, HIPFFT_BACKWARD);//����ȯ
	hipMemcpy(src, src_d, length*sizeof(float2), hipMemcpyDeviceToHost);
	for (int i = 0; i < length; i++)
		printf("%d, real: %f, imag: %f \n", i, src[i].x, src[i].y);

	hipFree(src_d);
}

int main()
{   
	hiprand();
	fft();
	nppFloatSum();
	//data_layout();
	//GPU - SIMD(T) :Single Instruction(function) Multi Data(Thread)
	//CPU - MIMD(T) :Multi  Instruction(function) Multi Data(Thread)
	//stream();
	//pinned_memory();
	
	//atomic_func();
	//cuda-memcheck ���ϸ�.exe
	//channel_mean();
	//device_query();
	//filter_2d();
	//filter_1d();
	//matrix_sum_by_row();
    return 0;// �������ϸ��Ϸ��� return 0 ���� ������ �մϴ�. 
}
