#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <malloc.h>

__global__ void mat_sum_by_row(
	float* dst, float*src,int stride){
	int tx = threadIdx.x;
	float sum = 0;
	for (int i = 0; i < stride; i++)	
		sum += src[tx + i * stride];	
	dst[tx] = sum;
}
__global__ void mat_sum_by_column(
	float* dst, float*src, int stride){
	int tx = threadIdx.x;
	float sum = 0;
	for (int i = 0; i < stride; i++)
		sum += src[tx * stride + i]; //���� : [tx + i * stride]
	dst[tx] = sum;
}
void matrix_sum_by_row(){
	int h = 4, w = 4;
	float *src_h, *dst_h;
	float *src_d, *dst_d;
	src_h = (float*)malloc(h*w*sizeof(float));
	dst_h = (float*)malloc(w*sizeof(float));
	hipMalloc(&src_d, h*w*sizeof(float));
	hipMalloc(&dst_d, w*sizeof(float));
	for (int i = 0; i < h*w; i++) src_h[i] = i;
	hipMemcpy(src_d, src_h, h*w*sizeof(float), hipMemcpyHostToDevice);
	mat_sum_by_column << <1, w >> > (dst_d, src_d, w);
	hipMemcpy(dst_h, dst_d, w*sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < w; i++)
		printf("%d %f\n", i, dst_h[i]);
}
__global__ void filter_1d_kernel(float*dst, float*src, float*filter, int f){
	int tx = threadIdx.x;//0,1,2,3
	//float sum = 0; // on-chip �ӵ� ����
	dst[tx] = 0;//0���� �ʱ�ȭ�� �ݵ�� �Ǿ��־�� �Ѵ�.
	for (int i = 0; i < f; i++)//f=3
	{
		//off-chip �ӵ� ���� 
		dst[tx] += src[tx + i] * filter[i];// 0��������� (0,1,2), 1��������� (1,2,3)...
	}
	//dst[tx] = sum;
}
void filter_1d(){
	int w = 6, f = 3;
	int out_length = w - (f / 2) * 2;// 4
	float *src_h, *filter_h, *dst_h;
	float *src_d, *filter_d, *dst_d;
	src_h = (float*)malloc(w*sizeof(float));
	filter_h = (float*)malloc(f*sizeof(float));
	dst_h = (float*)malloc(out_length*sizeof(float));
	hipMalloc(&src_d, w*sizeof(float));
	hipMalloc(&filter_d, f*sizeof(float));
	hipMalloc(&dst_d, out_length*sizeof(float));

	for (int i = 0; i < w; i++) src_h[i] = i;//�Է� ��ȣ
	for (int i = 0; i < f; i++) filter_h[i] = 1;//���� ��� 
	hipMemcpy(src_d, src_h, w*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(filter_d, filter_h, f*sizeof(float), hipMemcpyHostToDevice);
	filter_1d_kernel <<<1, out_length>>>(dst_d, src_d, filter_d, f);
	hipMemcpy(dst_h, dst_d, out_length*sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < out_length; i++)
		printf("%d %f\n", i, dst_h[i]);
}
// f=3, w=6, out_w=4
__global__ void filter_2d_kernel(float*dst, float*src, float*filter, int f, int w, int out_w){
	int bx = blockIdx.x; //0, 1
	int tx = threadIdx.x;//0,1,2,3
	float sum = 0;
	for (int i = 0; i < f; i++)//f=3
	{	
		// 0���/0������� (0,1,2), 1���/0��������� (6,7,8)...
		sum += src[bx * w + tx + i] * filter[i];
	}
	// 0���/0������� (0)�� ���� ����, 1���/0��������� (4) �� ���� ����Ѵ�
	dst[bx * out_w + tx] = sum;
}
void filter_2d(){
	int w = 6, h = 2, f = 3;//��ȣ�� ���� 6, ��ȣ�� ���� 2�� 
	int out_length = h * (w - (f / 2) * 2);// 2 * 4
	float *src_h, *filter_h, *dst_h;
	float *src_d, *filter_d, *dst_d;
	src_h = (float*)malloc(h * w*sizeof(float));
	filter_h = (float*)malloc(f*sizeof(float));
	dst_h = (float*)malloc(out_length*sizeof(float));
	hipMalloc(&src_d, h * w*sizeof(float));
	hipMalloc(&filter_d, f*sizeof(float));
	hipMalloc(&dst_d, out_length*sizeof(float));

	for (int i = 0; i < h * w; i++) src_h[i] = i;//�Է� ��ȣ
	for (int i = 0; i < f; i++) filter_h[i] = 1;//���� ��� 
	hipMemcpy(src_d, src_h, h * w*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(filter_d, filter_h, f*sizeof(float), hipMemcpyHostToDevice);
	filter_2d_kernel <<<2, 4 >>>(dst_d, src_d, filter_d, f, w, 4);
	hipMemcpy(dst_h, dst_d, out_length*sizeof(float), hipMemcpyDeviceToHost);
	for (int i = 0; i < out_length; i++)
		printf("%d %f\n", i, dst_h[i]);
}

int main()
{   
	filter_2d();
	//filter_1d();
	//matrix_sum_by_row();
    return 0;
}
