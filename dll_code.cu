
#include "hip/hip_runtime.h"

#include <stdio.h>

int *src_d;
int *dst_d;

extern "C" __declspec(dllexport) void init(int size, int *src_h){
	hipMalloc(&src_d, size * sizeof(int));
	hipMalloc(&dst_d, size * sizeof(int));
	hipHostRegister(&src_h, size * sizeof(int), hipHostRegisterDefault);
}

__global__ void kernel(int *dst, int* src){
	int tx = threadIdx.x;
	dst[tx] = src[tx] + 100;
}

extern "C" __declspec(dllexport) void add(int *dst, int* src, int size){
	//src = {1,2,3,4,5}, size = 5	
	hipMemcpy(src_d, src, size * sizeof(int), hipMemcpyHostToDevice);
	kernel <<<1, size >>>(dst_d, src_d);
	hipMemcpy(dst, dst_d, size * sizeof(int), hipMemcpyDeviceToHost);
}
