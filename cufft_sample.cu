#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hipfft/hipfft.h>
#include <malloc.h>
int main()
{
	//cufft.lib ��Ŀ �߰�
	int length = 1000;
	float2 * src = (float2*) malloc(length*sizeof(float2));
	for (int i = 0; i < length; i++)
	{
		src[i].x = i;//�Ǽ�
		src[i].y = 0;//���
	}
	float2 *src_d;
	hipMalloc(&src_d, length*sizeof(float2));
	hipMemcpy(src_d, src, length*sizeof(float2), hipMemcpyHostToDevice);

	hipfftHandle plan;
    hipfftPlan1d(&plan, length, HIPFFT_C2C, 1);//�Ķ���� ����
	hipfftExecC2C(plan, src_d, src_d, HIPFFT_BACKWARD);//����ȯ
	hipMemcpy(src, src_d, length*sizeof(float2), hipMemcpyDeviceToHost);
	for (int i = 0; i < length; i++)
		printf("%d, real: %f, imag: %f \n", i, src[i].x, src[i].y);
	
	hipFree(src_d);
    return 0;
}
