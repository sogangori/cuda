#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hipfft/hipfft.h>
#include <malloc.h>
int main()
{
	//cufft.lib 링커 추가
	int length = 1000;
	float2 * src = (float2*) malloc(length*sizeof(float2));
	for (int i = 0; i < length; i++)
	{
		src[i].x = i;//실수
		src[i].y = 0;//허수
	}
	float2 *src_d;
	hipMalloc(&src_d, length*sizeof(float2));
	hipMemcpy(src_d, src, length*sizeof(float2), hipMemcpyHostToDevice);

	hipfftHandle plan;
    hipfftPlan1d(&plan, length, HIPFFT_C2C, 1);//파라미터 셋팅
	hipfftExecC2C(plan, src_d, src_d, HIPFFT_BACKWARD);//정변환
	hipMemcpy(src, src_d, length*sizeof(float2), hipMemcpyDeviceToHost);
	for (int i = 0; i < length; i++)
		printf("%d, real: %f, imag: %f \n", i, src[i].x, src[i].y);
	
	hipFree(src_d);
    return 0;
}
