#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <malloc.h>

__global__ void kernel(short* dst, hipTextureObject_t tex) {

 int y = blockIdx.x;
 int x = threadIdx.x;
 int w = blockDim.x;
 int h = gridDim.x; 

 short v = tex2D<short>(tex, x, y);
 dst[y * w + x] = v * 10;
}

int main()
{
 int w = 4;
 int h = 4;
 int N = w*h;
 short * src = new short[N];
 short * out = new short[N];
 for (int i = 0; i < N; i++)
 {
  src[i] = i+1;
 }

 short *out_d;
 hipMalloc(&out_d, N*sizeof(short));
 // Allocate CUDA array in device memory
 hipChannelFormatDesc channelDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindSigned);//hipChannelFormatKindSigned,hipChannelFormatKindFloat
 hipArray* cuArray;
 hipMallocArray(&cuArray, &channelDesc, w, h); 
 hipMemcpyToArray(cuArray, 0, 0, src, N*sizeof(short), hipMemcpyHostToDevice);
 // create texture object
 hipResourceDesc resDesc;
 memset(&resDesc, 0, sizeof(resDesc));
 resDesc.resType = hipResourceTypeArray; //hipResourceTypeArray,hipResourceTypeLinear,hipResourceTypePitch2D
 resDesc.res.array.array = cuArray; 

 hipTextureDesc texDesc;
 memset(&texDesc, 0, sizeof(texDesc));
 texDesc.addressMode[0] = hipAddressModeWrap;//hipAddressModeWrap, hipAddressModeClamp
 texDesc.addressMode[1] = hipAddressModeWrap;
 texDesc.filterMode = hipFilterModePoint;//hipFilterModePoint, hipFilterModeLinear
 texDesc.readMode = hipReadModeElementType;//hipReadModeElementType,hipReadModeNormalizedFloat

 // create texture object: we only have to do this once!
 hipTextureObject_t tex = 0;
 hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

 kernel << <h, w >> >(out_d, tex);
 hipMemcpy(out, out_d, N*sizeof(short), hipMemcpyDeviceToHost);
 for (int i = 0; i < N; i++)
 {
  printf("%d %d \n", i, out[i]);
 }
 // destroy texture object
 hipDestroyTextureObject(tex);
 return 0;
}